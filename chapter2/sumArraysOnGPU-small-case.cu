#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define CHECK(call)                                                             \
{                                                                               \
    const hipError_t error = call;                                             \
    if (error != hipSuccess){                                                  \
        printf("Error: %s: %d, ", __FILE__, __LINE__);                          \
        printf("code: %d, reason: %s \n", error, hipGetErrorString(error));    \
        exit(1);                                                                \
    }                                                                           \
}

void  checkResult(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1e-8;
    bool match = 1; 
    for(int i=0; i<N; i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon) {
            match = 0; 
            printf("Arrays do not match! \n");
            printf("host %5.2f gpu %5.2fat current %d \n", hostRef[i], gpuRef[i], i);
            break;
        }
    }
    if (match) { 
        printf("Arrays match! \n");
    }
}

void sumArraysOnHost(float *A, float *B, float *C, const int N){
    // Do elementwise add
    for(int idx=0; idx<N; idx++){
        C[idx] = A[idx] + B[idx];
    }
}

__global__ void sumArraysOnGPU(float *A, float *B, float *C){
    // Do elementwise add
    int idx = threadIdx.x;
    // int idx = blockIdx.x*blockDim.x + threadIdx.x;
    C[idx] = A[idx] + B[idx];
}

void initialData(float *ip, int size){
    // Generate different seed for random number
    time_t t; 
    srand((unsigned int) time(&t));
    for(int i=0; i<size; i++){
        ip[i] = (float)(rand() &0xFF) /10.0f; 
    }
}

int main(){
    printf("starting... \n");

    // set up device 
    int dev = 0; 
    hipSetDevice(dev);

    // set up data size of vectors
    int nElem = 32; 
    size_t nBytes = nElem * sizeof(float);
    float *h_A, *h_B, *hostRef, *gpuRef;
    h_A = (float *)malloc(nBytes);
    h_B = (float *)malloc(nBytes);
    hostRef = (float *)malloc(nBytes);
    gpuRef = (float *)malloc(nBytes);

    // initial data
    initialData(h_A, nElem);
    initialData(h_B, nElem);

    memset(hostRef, 0, nBytes);
    memset(gpuRef, 0, nBytes);


    float *d_A, *d_B, *d_C;
    hipMalloc((float **)&d_A, nBytes);
    hipMalloc((float **)&d_B, nBytes);
    hipMalloc((float **)&d_C, nBytes);

    hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);

    // invoke kernel at host side
    dim3 block(nElem);
    dim3 grid(nElem/block.x);

    sumArraysOnGPU<<<grid, block>>>(d_A, d_B, d_C);
    printf("Execution configuration <<<%d %d>>> \n", grid.x, block.x);
    
    // copy kernel result back to host side
    hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);

    // add vector at host side for result
    sumArraysOnHost(h_A, h_B, hostRef, nElem);

    // check device results
    checkResult(hostRef, gpuRef, nElem);

    // free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    //free host memory
    free(h_A);
    free(h_B);
    free(gpuRef);
    free(hostRef);
    return 0;
}
