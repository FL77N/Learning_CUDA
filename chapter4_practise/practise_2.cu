#include "hip/hip_runtime.h"
#include "stdio.h"

__device__ float devData; 

__global__ void checkGlobalVariable(){
    printf("Device: the value of the global variable is: %f\n", devData);
    devData += 2.0f;
}

int main(void){
    float value = 3.14f; 
    float *dptr = NULL; 
    hipGetSymbolAddress((void **)&dptr, HIP_SYMBOL(devData));
    hipMemcpy(dptr, &value, sizeof(float), hipMemcpyHostToDevice);
    printf("Host: copied %f to the global variable \n", value);

    checkGlobalVariable<<<1, 1>>>();

    // copy back 
    hipMemcpy(&value, dptr, sizeof(float), hipMemcpyDeviceToHost);
    printf("Host the value changed by the kernel to %f \n", value);

    hipDeviceReset(); 
    return EXIT_SUCCESS; 

}