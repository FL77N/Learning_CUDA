#include "hip/hip_runtime.h"
#include "../common/common.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define BDIMX 32 
#define BDIMY 32
#define IPAD 1

void printData(char *msg, int *in, const int size){
    printf("%s: ", msg);
    for(int i=0; i < size; i++){
        printf("%5d", in[i]);
        fflush(stdout);
    }
    printf("\n");
    return;
}

__global__ void setRowReadRow(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.x][threadIdx.y] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadColDyn(int *out){
    extern __shared__ int tile[];

    unsigned int row_idx = threadIdx.y * blockDim.x + threadIdx.x;
    unsigned int col_idx = threadIdx.x * blockDim.y + threadIdx.y;

    tile[row_idx] = row_idx;
    __syncthreads();
    out[row_idx] = tile[col_idx];
}

__global__ void setRowReadColPad(int *out){
    __shared__ int tile[BDIMY][BDIMX+IPAD];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx;

    __syncthreads();
    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setRowReadColDynPad(int *out){
    extern __shared__ int tile[];

    unsigned int row_idx = threadIdx.y * (blockDim.x + 1) + threadIdx.x;
    unsigned int col_idx = threadIdx.x * (blockDim.x + 1) + threadIdx.y;

    unsigned int g_idx = threadIdx.y * blockDim.x + threadIdx.x;

    tile[row_idx] = g_idx;
    __syncthreads();
    out[g_idx] = tile[col_idx];
}

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CHECK(hipDeviceGetSharedMemConfig(&pConfig));
    printf("with Bank Mode: %s ", pConfig == 1 ? "4-Byte": "8-Byte");

    int nx = BDIMX; 
    int ny = BDIMY; 
    bool iprintf = 0;

    if(argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    dim3 block(BDIMX, BDIMY);
    dim3 grid(1, 1); 
    printf("<<< grid (%d, %d) block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

    int *d_C;
    CHECK(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef = (int *)malloc(nBytes);

    CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C); 
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf) printData("set col read col: ", gpuRef, nx*ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C); 
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf) printData("set row read row: ", gpuRef, nx*ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf)  printData("set row read col: ", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDyn<<<grid, block, BDIMX*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf)  printData("set row read col dyn", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColPad<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf)  printData("set row read col pad", gpuRef, nx * ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadColDynPad<<<grid, block, (BDIMX + IPAD)*BDIMY*sizeof(int)>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf)  printData("set row read col DP ", gpuRef, nx * ny);

    CHECK(hipFree(d_C)); 
    free(gpuRef);
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}