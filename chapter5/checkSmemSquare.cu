#include "hip/hip_runtime.h"
#include "../common/common.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define BDIMX 32 
#define BDIMY 32
#define IPAD 1

void printData(char *msg, int *in, const int size){
    printf("%s: ", msg);
    for(int i=0; i < size; i++){
        printf("%5d", in[i]);
        fflush(stdout);
    }
    printf("\n");
    return;
}

__global__ void setRowReadRow(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setColReadCol(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.x][threadIdx.y] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setRowReadCol(int *out){
    __shared__ int tile[BDIMY][BDIMX];
    unsigned int idx = threadIdx.y * blockDim.x + threadIdx.x;
    tile[threadIdx.y][threadIdx.x] = idx; 

    __syncthreads();

    out[idx] = tile[threadIdx.x][threadIdx.y];
}

int main(int argc, char** argv){
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s at ", argv[0]);
    printf("device %d: %s", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    hipSharedMemConfig pConfig;
    CHECK(hipDeviceGetSharedMemConfig(&pConfig));
    printf("with Bank Mode: %s ", pConfig == 1 ? "4-Byte": "8-Byte");

    int nx = BDIMX; 
    int ny = BDIMY; 
    bool iprintf = 0;

    if(argc > 1) iprintf = atoi(argv[1]);

    size_t nBytes = nx * ny * sizeof(int);

    dim3 block(BDIMX, BDIMY);
    dim3 grid(1, 1); 
    printf("<<< grid (%d, %d) block (%d, %d)>>>\n", grid.x, grid.y, block.x, block.y);

    int *d_C;
    CHECK(hipMalloc((int**)&d_C, nBytes));
    int *gpuRef = (int *)malloc(nBytes);

    CHECK(hipMemset(d_C, 0, nBytes));
    setColReadCol<<<grid, block>>>(d_C); 
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf) printData("set col read col: ", gpuRef, nx*ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadRow<<<grid, block>>>(d_C); 
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf) printData("set row read row: ", gpuRef, nx*ny);

    CHECK(hipMemset(d_C, 0, nBytes));
    setRowReadCol<<<grid, block>>>(d_C);
    CHECK(hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost));
    if(iprintf)  printData("set row read col: ", gpuRef, nx * ny);

    CHECK(hipFree(d_C)); 
    free(gpuRef);
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}